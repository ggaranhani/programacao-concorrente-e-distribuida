#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
//#include <sys/time.h>

int MAX_THREADS;
#define tam 1.0
#define dx 0.00001
#define dt 0.000001
#define T 0.01
#define kappa 0.000045

typedef long long int ind;

__global__ void calculo(double *u, double *prev, double k, double t, double x, int n) {
    ind i =  blockDim.x * blockIdx.x + threadIdx.x;
    if (i > 0 && i < (n - 1))
        u[i] = prev[i] + k * t / (x * x) * (prev[i-1] - 2 * prev[i] + prev[i+1]);
}

__global__ void contorno(double *u, int n) {
    ind i =  blockDim.x * blockIdx.x + threadIdx.x;
    if(i == 1) 
	u[0] = u[n] = 0.0;
}

int main(int argc, char **argv) {
  if(argc >= 2){
      MAX_THREADS = atoi(argv[1]);
  }
  else{
      MAX_THREADS = 64;
  }
  printf("INICIANDO COM %d THREADS\n", MAX_THREADS);
  double *tmp, *u, *u_prev;
  double x = dx, t;
  ind i, n, maxloc, size, num_blocos;
  double *d_u, *d_u_prev;
  float tempo = 0.0;
  hipEvent_t start, stop;
  /* Calculando quantidade de pontos */
  n = tam / dx;
  num_blocos = ceil((n+1)/MAX_THREADS);

  /* Alocando vetores */
  u = (double *) malloc((n+1)*sizeof(double));
  u_prev = (double *) malloc((n+1)*sizeof(double));

  size = (n + 1) * sizeof(ind);
  hipMalloc((void **) &d_u, size);
  hipMalloc((void **) &d_u_prev, size);

  printf("Inicio: qtde=%ld, dt=%g, dx=%g, dx²=%g, kappa=%f, const=%f\n",
	 (n+1), dt, dx, dx*dx, kappa, kappa*dt/(dx*dx));
  printf("Iteracoes previstas: %g\n", T/dt);

  x = 0;
  for (i = 0; i < n+1; i++) {
      if (x <= 0.5)
          u_prev[i] = 200 * x;
      else
          u_prev[i] = 200 * (1.-x);
      x += dx;
  }

  printf("dx=%g, x=%g, x-dx=%g\n", dx, x, x-dx);
  printf("u_prev[0,1]=%g, %g\n",u_prev[0],u_prev[1]);
  printf("u_prev[n-1,n]=%g, %g\n",u_prev[n-1],u_prev[n]);

  hipMemcpy(d_u_prev, u_prev, size, hipMemcpyHostToDevice);

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  t = 0.;
  while (t < T) {
	x = dx;
	calculo <<<num_blocos, MAX_THREADS>>> (d_u, d_u_prev, kappa, dt, dx, n+1);
	hipDeviceSynchronize();
	contorno <<<num_blocos, MAX_THREADS>>> (d_u, n);
	tmp = d_u_prev; 
	d_u_prev = d_u;
	d_u = tmp;
	t += dt;
  }

  hipMemcpy(u, d_u, size, hipMemcpyDeviceToHost);

  /* Calculando o maior valor e sua localizacao */
  maxloc = 0;
  for (i = 1; i < n+1; i++)
      if (u[i] > u[maxloc]) maxloc = i;

  printf("Maior valor u[%ld] = %g\n", maxloc, u[maxloc]);

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&tempo, start, stop);
  printf("Tempo total: %f ms\n", tempo);
  hipEventDestroy(start);
        hipEventDestroy(stop);

  free(u);
  free(u_prev);
  hipFree(d_u);
  hipFree(d_u_prev);

  return 0;
}
